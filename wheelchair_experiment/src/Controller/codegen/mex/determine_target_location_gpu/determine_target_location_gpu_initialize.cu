//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// determine_target_location_gpu_initialize.cu
//
// Code generation for function 'determine_target_location_gpu_initialize'
//

// Include files
#include "determine_target_location_gpu_initialize.h"
#include "_coder_determine_target_location_gpu_mex.h"
#include "determine_target_location_gpu_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void determine_target_location_gpu_initialize()
{
  mex_InitInfAndNan();
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

// End of code generation (determine_target_location_gpu_initialize.cu)
