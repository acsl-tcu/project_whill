#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// determine_target_location_gpu.cu
//
// Code generation for function 'determine_target_location_gpu'
//

// Include files
#include "determine_target_location_gpu.h"
#include "determine_target_location_gpu_data.h"
#include "determine_target_location_gpu_emxutil.h"
#include "determine_target_location_gpu_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    6,                               // lineNo
    25,                              // colNo
    "determine_target_location_gpu", // fName
    "/home/student/Document/10\xe5\x8f\xb7\xe9\xa4\xa8"
    "4\xe9\x9a\x8e\xe9\x80\x9a\xe8\xb7\xaf\xe8\xb5\xb0\xe8\xa1\x8c/"
    "10\xe5\x8f\xb7"
    "\xe9\xa4\xa8"
    "4\xe9\x9a\x8e\xe9\x80\x9a\xe8\xb7\xaf\xe8\xb5\xb0\xe8\xa1\x8c_2025_07_02/"
    "capsuele_v5.3_\xe8\xa4\x87\xe6\x95\xb0\xe6"
    "\xad\xa9\xe8\xa1\x8c\xe8\x80\x85/src/Controller/"
    "determine_target_location_gpu.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    60,                                                             // lineNo
    20,                                                             // colNo
    "dot",                                                          // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/specfun/dot.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    6,                               // lineNo
    21,                              // colNo
    "determine_target_location_gpu", // fName
    "/home/student/Document/10\xe5\x8f\xb7\xe9\xa4\xa8"
    "4\xe9\x9a\x8e\xe9\x80\x9a\xe8\xb7\xaf\xe8\xb5\xb0\xe8\xa1\x8c/"
    "10\xe5\x8f\xb7"
    "\xe9\xa4\xa8"
    "4\xe9\x9a\x8e\xe9\x80\x9a\xe8\xb7\xaf\xe8\xb5\xb0\xe8\xa1\x8c_2025_07_02/"
    "capsuele_v5.3_\xe8\xa4\x87\xe6\x95\xb0\xe6"
    "\xad\xa9\xe8\xa1\x8c\xe8\x80\x85/src/Controller/"
    "determine_target_location_gpu.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    6,                               // lineNo
    31,                              // colNo
    "determine_target_location_gpu", // fName
    "/home/student/Document/10\xe5\x8f\xb7\xe9\xa4\xa8"
    "4\xe9\x9a\x8e\xe9\x80\x9a\xe8\xb7\xaf\xe8\xb5\xb0\xe8\xa1\x8c/"
    "10\xe5\x8f\xb7"
    "\xe9\xa4\xa8"
    "4\xe9\x9a\x8e\xe9\x80\x9a\xe8\xb7\xaf\xe8\xb5\xb0\xe8\xa1\x8c_2025_07_02/"
    "capsuele_v5.3_\xe8\xa4\x87\xe6\x95\xb0\xe6"
    "\xad\xa9\xe8\xa1\x8c\xe8\x80\x85/src/Controller/"
    "determine_target_location_gpu.m" // pName
};

// Function Declarations
static void binary_expand_op(real_T in1_data[], int32_T in1_size[2],
                             const struct0_T *in2,
                             const emxArray_boolean_T *in3,
                             const int32_T in4[2]);

static void checkCudaError(hipError_t errorCode, const char_T *file,
                           int32_T b_line);

static uint64_T computeNumIters(int32_T ub, int32_T b_ub);

static uint64_T computeNumIters(int32_T ub, int32_T b_ub, int32_T c_ub);

static uint64_T computeNumIters(int32_T ub);

static __global__ void
f_determine_target_location_gpu(const struct0_T *obj, const int32_T b_obj,
                                int32_T obj_data[210000]);

static __global__ void
g_determine_target_location_gpu(const struct0_T *obj,
                                const int32_T obj_data[210000], const int32_T b,
                                real_T dv_data[420000]);

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy);

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString, const char_T *file,
                          int32_T b_line);

static void gpuThrowError(const char_T *file, int32_T b_line);

static __global__ void h_determine_target_location_gpu(const struct0_T *obj,
                                                       const int32_T b,
                                                       real_T dv1_data[420000]);

static __global__ void
i_determine_target_location_gpu(const real_T dv_data[420000],
                                const real_T dv1_data[420000],
                                const int32_T obj, emxArray_real_T a);

static __global__ void j_determine_target_location_gpu(const real_T px_data[],
                                                       const int32_T px_size,
                                                       real_T dv_data[420000]);

static __global__ void
k_determine_target_location_gpu(const real_T dv1_data[420000],
                                const real_T dv_data[420000], const int32_T b,
                                emxArray_real_T b_b);

static __global__ void l_determine_target_location_gpu(const emxArray_real_T c,
                                                       const int32_T b_c,
                                                       emxArray_boolean_T a);

static __global__ void m_determine_target_location_gpu(
    const int32_T npages, const emxArray_boolean_T a, const struct0_T *obj,
    const int32_T b_obj, real_T tgt_n_data[]);

// Function Definitions
static void binary_expand_op(real_T in1_data[], int32_T in1_size[2],
                             const struct0_T *in2,
                             const emxArray_boolean_T *in3,
                             const int32_T in4[2])
{
  int32_T aux_0_1;
  int32_T aux_1_1;
  int32_T aux_2_1;
  int32_T b_in2;
  int32_T i;
  int32_T in4_idx_0;
  int32_T in4_idx_1;
  int32_T stride_0_1;
  int32_T stride_1_0;
  int32_T stride_1_1;
  int32_T stride_2_1;
  in4_idx_0 = in4[0];
  in4_idx_1 = in4[1];
  b_in2 = in2->waypoint.size[0];
  in1_size[0] = 21;
  if (in2->target_n.size[1] == 1) {
    i = in4_idx_1;
  } else {
    i = in2->target_n.size[1];
  }
  if (i == 1) {
    in1_size[1] = in2->target_n.size[1];
  } else if (in2->target_n.size[1] == 1) {
    in1_size[1] = in4_idx_1;
  } else {
    in1_size[1] = in2->target_n.size[1];
  }
  stride_0_1 = (in2->target_n.size[1] != 1);
  stride_1_0 = (in4_idx_0 != 1);
  stride_1_1 = (in4_idx_1 != 1);
  stride_2_1 = (in2->target_n.size[1] != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  aux_2_1 = 0;
  if (in2->target_n.size[1] == 1) {
    i = in4_idx_1;
  } else {
    i = in2->target_n.size[1];
  }
  if (i == 1) {
    in4_idx_1 = in2->target_n.size[1];
  } else if (in2->target_n.size[1] != 1) {
    in4_idx_1 = in2->target_n.size[1];
  }
  for (i = 0; i < in4_idx_1; i++) {
    for (int32_T i1{0}; i1 < 21; i1++) {
      in1_data[i1 + 21 * i] =
          in2->target_n.data[i1 + 21 * aux_0_1] +
          static_cast<real_T>(
              in3->data[i1 * stride_1_0 + in4_idx_0 * aux_1_1] &&
              (in2->target_n.data[i1 + 21 * aux_2_1] != b_in2));
    }
    aux_2_1 += stride_2_1;
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
}

static void checkCudaError(hipError_t errorCode, const char_T *file,
                           int32_T b_line)
{
  if (errorCode != hipSuccess) {
    gpuThrowError(errorCode, hipGetErrorName(errorCode),
                  hipGetErrorString(errorCode), file, b_line);
  }
}

static uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  boolean_T overflow;
  overflow = false;
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
    overflow = (numIters > MAX_uint64_T / static_cast<uint64_T>(b_ub + 1));
  }
  numIters *= n;
  if (overflow) {
    gpuThrowError(__FILE__, __LINE__);
  }
  return numIters;
}

static uint64_T computeNumIters(int32_T ub, int32_T b_ub, int32_T c_ub)
{
  uint64_T n;
  uint64_T numIters;
  boolean_T overflow;
  overflow = false;
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
    overflow = (numIters > MAX_uint64_T / static_cast<uint64_T>(b_ub + 1));
  }
  numIters *= n;
  n = 0UL;
  if (c_ub >= 0) {
    n = static_cast<uint64_T>(c_ub + 1);
    overflow = ((numIters > MAX_uint64_T / static_cast<uint64_T>(c_ub + 1)) ||
                overflow);
  }
  numIters *= n;
  if (overflow) {
    gpuThrowError(__FILE__, __LINE__);
  }
  return numIters;
}

static uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  return numIters;
}

static __global__
    __launch_bounds__(1024, 1) void f_determine_target_location_gpu(
        const struct0_T *obj, const int32_T b_obj, int32_T obj_data[210000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_obj);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    obj_data[i] = static_cast<int32_T>(obj->target_n.data[i] - 1.0);
  }
}

static __global__
    __launch_bounds__(1024, 1) void g_determine_target_location_gpu(
        const struct0_T *obj, const int32_T obj_data[210000], const int32_T b,
        real_T dv_data[420000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b) + 1UL) * 2UL - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T i1;
    i1 = static_cast<int32_T>(idx % 2UL);
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(i1)) / 2UL);
    dv_data[i1 + 2 * i] =
        obj->waypoint.data[(obj_data[i] + obj->waypoint.size[0] * i1) - 1];
  }
}

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  boolean_T *newData;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(boolean_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(boolean_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu)
{
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  if (cpu->data) {
    checkCudaError(
        hipMemcpy(gpu->data, cpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(real_T),
                   hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
}

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  if (gpu->data) {
    checkCudaError(
        hipMemcpy(cpu->data, gpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(boolean_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  if (gpu->data) {
    checkCudaError(
        hipMemcpy(cpu->data, gpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(real_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
}

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_boolean_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

static void gpuThrowError(const char_T *file, int32_T b_line)
{
  emlrtRTEInfo rtInfo;
  rtInfo.lineNo = b_line;
  rtInfo.colNo = 0;
  rtInfo.fName = "";
  rtInfo.pName = file;
  emlrtCUDAError(
      0U, (char_T *)"_",
      (char_T
           *)"Unable to launch kernel. Loop nest contains too many iterations.",
      &rtInfo, emlrtRootTLSGlobal);
}

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString, const char_T *file,
                          int32_T b_line)
{
  emlrtRTEInfo rtInfo;
  rtInfo.lineNo = b_line;
  rtInfo.colNo = 0;
  rtInfo.fName = "";
  rtInfo.pName = file;
  emlrtCUDAError(errorCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
}

static __global__
    __launch_bounds__(1024, 1) void h_determine_target_location_gpu(
        const struct0_T *obj, const int32_T b, real_T dv1_data[420000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b) + 1UL) * 2UL - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T i1;
    i1 = static_cast<int32_T>(idx % 2UL);
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(i1)) / 2UL);
    dv1_data[i1 + 2 * i] =
        obj->waypoint.data[(static_cast<int32_T>(obj->target_n.data[i]) +
                            obj->waypoint.size[0] * i1) -
                           1];
  }
}

static __global__ __launch_bounds__(
    1024, 1) void i_determine_target_location_gpu(const real_T dv_data[420000],
                                                  const real_T dv1_data[420000],
                                                  const int32_T obj,
                                                  emxArray_real_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(obj);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    a.data[i] = dv1_data[i] - dv_data[i];
  }
}

static __global__
    __launch_bounds__(1024, 1) void j_determine_target_location_gpu(
        const real_T px_data[], const int32_T px_size, real_T dv_data[420000])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(px_size) + 1UL) * 21UL * 2UL - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    uint64_T tmpIndex;
    int32_T i;
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % 2UL);
    tmpIndex = (idx - static_cast<uint64_T>(i2)) / 2UL;
    i1 = static_cast<int32_T>(tmpIndex % 21UL);
    tmpIndex = (tmpIndex - static_cast<uint64_T>(i1)) / 21UL;
    i = static_cast<int32_T>(tmpIndex);
    dv_data[(i2 + 2 * i1) + 42 * i] = px_data[(i2 + 3 * i1) + 63 * i];
  }
}

static __global__ __launch_bounds__(
    1024, 1) void k_determine_target_location_gpu(const real_T dv1_data[420000],
                                                  const real_T dv_data[420000],
                                                  const int32_T b,
                                                  emxArray_real_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_b.data[i] = dv_data[i] - dv1_data[i];
  }
}

static __global__
    __launch_bounds__(1024, 1) void l_determine_target_location_gpu(
        const emxArray_real_T c, const int32_T b_c, emxArray_boolean_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    a.data[i] = (c.data[i] >= 0.0);
  }
}

static __global__
    __launch_bounds__(1024, 1) void m_determine_target_location_gpu(
        const int32_T npages, const emxArray_boolean_T a, const struct0_T *obj,
        const int32_T b_obj, real_T tgt_n_data[])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_obj);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T d;
    int32_T i;
    i = static_cast<int32_T>(idx);
    d = obj->target_n.data[i];
    tgt_n_data[i] = d + static_cast<real_T>(a.data[i] &&
                                            (d != static_cast<real_T>(npages)));
  }
}

void determine_target_location_gpu(const struct0_T *cpu_obj,
                                   const real_T cpu_px_data[],
                                   const int32_T px_size[3],
                                   real_T cpu_tgt_n_data[],
                                   int32_T tgt_n_size[2])
{
  dim3 block;
  dim3 grid;
  emxArray_boolean_T b_gpu_a;
  emxArray_boolean_T *b_cpu_a;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_b;
  emxArray_real_T gpu_c;
  emxArray_real_T *cpu_a;
  emxArray_real_T *cpu_b;
  emxArray_real_T *cpu_c;
  struct0_T *gpu_obj;
  real_T(*gpu_dv1_data)[420000];
  real_T(*gpu_dv_data)[420000];
  real_T *gpu_px_data;
  real_T *gpu_tgt_n_data;
  uint64_T numIters;
  int32_T(*gpu_obj_data)[210000];
  int32_T szb[2];
  int32_T i2;
  int32_T npages;
  boolean_T a_outdatedOnCpu;
  boolean_T b_outdatedOnCpu;
  boolean_T c_outdatedOnGpu;
  boolean_T validLaunchParams;
  gpuEmxReset_boolean_T(&b_gpu_a);
  gpuEmxReset_real_T(&gpu_c);
  gpuEmxReset_real_T(&gpu_b);
  gpuEmxReset_real_T(&gpu_a);
  checkCudaError(mwCudaMalloc(&gpu_dv1_data, 3360000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_dv_data, 3360000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_obj_data, 840000UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_tgt_n_data, 8820000U * sizeof(real_T)),
                 __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_px_data, 630000U * sizeof(real_T)), __FILE__,
                 __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_obj, 1680832UL), __FILE__, __LINE__);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  waypointの更新
  i2 = 21 * cpu_obj->target_n.size[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(i2), &grid, &block, 2147483647U);
  checkCudaError(
      hipMemcpy(gpu_obj, cpu_obj, 1680832UL, hipMemcpyHostToDevice), __FILE__,
      __LINE__);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    f_determine_target_location_gpu<<<grid, block>>>(gpu_obj, i2,
                                                     *gpu_obj_data);
  }
  i2 = 21 * cpu_obj->target_n.size[1];
  numIters = computeNumIters(i2 - 1, 1);
  mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    g_determine_target_location_gpu<<<grid, block>>>(gpu_obj, *gpu_obj_data,
                                                     i2 - 1, *gpu_dv_data);
  }
  i2 = 21 * cpu_obj->target_n.size[1];
  numIters = computeNumIters(i2 - 1, 1);
  mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    h_determine_target_location_gpu<<<grid, block>>>(gpu_obj, i2 - 1,
                                                     *gpu_dv1_data);
  }
  emxInit_real_T(&cpu_a, 4, &emlrtRTEI, true);
  npages = cpu_a->size[0] * cpu_a->size[1] * cpu_a->size[2] * cpu_a->size[3];
  cpu_a->size[0] = 1;
  cpu_a->size[1] = 2;
  cpu_a->size[2] = static_cast<int32_T>(cpu_obj->K);
  cpu_a->size[3] = static_cast<int32_T>(cpu_obj->NP);
  emxEnsureCapacity_real_T(cpu_a, npages, &emlrtRTEI);
  i2 =
      2 * static_cast<int32_T>(cpu_obj->K) * static_cast<int32_T>(cpu_obj->NP) -
      1;
  mwGetLaunchParameters1D(computeNumIters(i2), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    i_determine_target_location_gpu<<<grid, block>>>(*gpu_dv_data,
                                                     *gpu_dv1_data, i2, gpu_a);
  }
  a_outdatedOnCpu = true;
  emxInit_real_T(&cpu_b, 4, &d_emlrtRTEI, true);
  numIters = computeNumIters(px_size[2] - 1, 20, 1);
  mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
  checkCudaError(
      hipMemcpy(gpu_px_data, cpu_px_data,
                 static_cast<uint32_T>(3 * (21 * px_size[2])) * sizeof(real_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    j_determine_target_location_gpu<<<grid, block>>>(
        gpu_px_data, px_size[2] - 1, *gpu_dv_data);
  }
  npages = cpu_b->size[0] * cpu_b->size[1] * cpu_b->size[2] * cpu_b->size[3];
  cpu_b->size[0] = 1;
  cpu_b->size[1] = 2;
  cpu_b->size[2] = static_cast<int32_T>(cpu_obj->K);
  cpu_b->size[3] = static_cast<int32_T>(cpu_obj->NP);
  emxEnsureCapacity_real_T(cpu_b, npages, &d_emlrtRTEI);
  i2 =
      2 * static_cast<int32_T>(cpu_obj->K) * static_cast<int32_T>(cpu_obj->NP) -
      1;
  mwGetLaunchParameters1D(computeNumIters(i2), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    k_determine_target_location_gpu<<<grid, block>>>(*gpu_dv1_data,
                                                     *gpu_dv_data, i2, gpu_b);
  }
  b_outdatedOnCpu = true;
  emxInit_real_T(&cpu_c, 4, &c_emlrtRTEI, true);
  c_outdatedOnGpu = false;
  npages = cpu_c->size[0] * cpu_c->size[1] * cpu_c->size[2] * cpu_c->size[3];
  cpu_c->size[0] = 1;
  cpu_c->size[1] = 1;
  cpu_c->size[2] = cpu_a->size[2];
  cpu_c->size[3] = cpu_a->size[3];
  emxEnsureCapacity_real_T(cpu_c, npages, &b_emlrtRTEI);
  npages = cpu_a->size[2] * cpu_a->size[3];
  i2 = 1;
  for (int32_T px_size_idx_2{0}; px_size_idx_2 < npages; px_size_idx_2++) {
    int32_T i1;
    i1 = i2 - 1;
    i2 += 2;
    if (a_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
    }
    a_outdatedOnCpu = false;
    if (b_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real_T(cpu_b, &gpu_b);
    }
    b_outdatedOnCpu = false;
    cpu_c->data[px_size_idx_2] = cpu_a->data[i1] * cpu_b->data[i1] +
                                 cpu_a->data[i1 + 1] * cpu_b->data[i1 + 1];
    c_outdatedOnGpu = true;
  }
  emxFree_real_T(&cpu_b);
  emxFree_real_T(&cpu_a);
  emxInit_boolean_T(&b_cpu_a, 4, &c_emlrtRTEI, true);
  npages =
      b_cpu_a->size[0] * b_cpu_a->size[1] * b_cpu_a->size[2] * b_cpu_a->size[3];
  b_cpu_a->size[0] = 1;
  b_cpu_a->size[1] = 1;
  b_cpu_a->size[2] = cpu_c->size[2];
  b_cpu_a->size[3] = cpu_c->size[3];
  emxEnsureCapacity_boolean_T(b_cpu_a, npages, &c_emlrtRTEI);
  i2 = cpu_c->size[2] * cpu_c->size[3] - 1;
  mwGetLaunchParameters1D(computeNumIters(i2), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_c, &gpu_c, !c_outdatedOnGpu);
  gpuEmxEnsureCapacity_boolean_T(b_cpu_a, &b_gpu_a, true);
  if (c_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_c, cpu_c);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    l_determine_target_location_gpu<<<grid, block>>>(gpu_c, i2, b_gpu_a);
  }
  emxFree_real_T(&cpu_c);
  szb[0] = 1;
  szb[1] = 1;
  a_outdatedOnCpu = (b_cpu_a->size[2] == 1);
  if ((!a_outdatedOnCpu) || (b_cpu_a->size[3] != 1)) {
    a_outdatedOnCpu = false;
  }
  if (!a_outdatedOnCpu) {
    npages = 0;
    if (b_cpu_a->size[2] != 1) {
      npages = 1;
      szb[0] = b_cpu_a->size[2];
    }
    if (b_cpu_a->size[3] != 1) {
      szb[npages] = b_cpu_a->size[3];
    }
  }
  if (szb[1] == 1) {
    i2 = cpu_obj->target_n.size[1];
  } else {
    i2 = szb[1];
  }
  if ((szb[0] == 21) && (szb[1] == cpu_obj->target_n.size[1]) &&
      (cpu_obj->target_n.size[1] == i2)) {
    tgt_n_size[0] = 21;
    tgt_n_size[1] = cpu_obj->target_n.size[1];
    i2 = 21 * cpu_obj->target_n.size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i2), &grid, &block, 2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      m_determine_target_location_gpu<<<grid, block>>>(
          cpu_obj->waypoint.size[0], b_gpu_a, gpu_obj, i2, gpu_tgt_n_data);
    }
    a_outdatedOnCpu = true;
  } else {
    gpuEmxMemcpyGpuToCpu_boolean_T(b_cpu_a, &b_gpu_a);
    binary_expand_op(cpu_tgt_n_data, tgt_n_size, cpu_obj, b_cpu_a, szb);
    a_outdatedOnCpu = false;
  }
  emxFree_boolean_T(&b_cpu_a);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (a_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(cpu_tgt_n_data, gpu_tgt_n_data,
                   static_cast<uint32_T>(21 * tgt_n_size[1]) * sizeof(real_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  checkCudaError(mwCudaFree(gpu_obj), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_px_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_tgt_n_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_obj_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_dv_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_dv1_data), __FILE__, __LINE__);
  gpuEmxFree_real_T(&gpu_a);
  gpuEmxFree_real_T(&gpu_b);
  gpuEmxFree_real_T(&gpu_c);
  gpuEmxFree_boolean_T(&b_gpu_a);
}

// End of code generation (determine_target_location_gpu.cu)
