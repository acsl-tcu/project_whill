//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// determine_target_location_gpu_terminate.cu
//
// Code generation for function 'determine_target_location_gpu_terminate'
//

// Include files
#include "determine_target_location_gpu_terminate.h"
#include "_coder_determine_target_location_gpu_mex.h"
#include "determine_target_location_gpu_data.h"
#include "rt_nonfinite.h"

// Function Declarations
static void checkCudaError(hipError_t errorCode);

static void emlrtExitTimeCleanupDtorFcn(const void *r);

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString);

// Function Definitions
static void checkCudaError(hipError_t errorCode)
{
  if (errorCode != hipSuccess) {
    gpuThrowError(errorCode, hipGetErrorName(errorCode),
                  hipGetErrorString(errorCode));
  }
}

static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString)
{
  emlrtThinCUDAError(errorCode, (char_T *)errorName, (char_T *)errorString,
                     (char_T *)"SafeBuild", emlrtRootTLSGlobal);
}

void determine_target_location_gpu_atexit()
{
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
}

void determine_target_location_gpu_terminate()
{
  checkCudaError(hipGetLastError());
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (determine_target_location_gpu_terminate.cu)
