//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// EvaluationPath_GPU_initialize.cu
//
// Code generation for function 'EvaluationPath_GPU_initialize'
//

// Include files
#include "EvaluationPath_GPU_initialize.h"
#include "EvaluationPath_GPU_data.h"
#include "_coder_EvaluationPath_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void EvaluationPath_GPU_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (EvaluationPath_GPU_initialize.cu)
