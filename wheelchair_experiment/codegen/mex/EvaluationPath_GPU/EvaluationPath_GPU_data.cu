//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// EvaluationPath_GPU_data.cu
//
// Code generation for function 'EvaluationPath_GPU_data'
//

// Include files
#include "EvaluationPath_GPU_data.h"
#include "rt_nonfinite.h"

// Variable Definitions
emlrtCTX emlrtRootTLSGlobal{nullptr};

emlrtContext emlrtContextGlobal{
    true,                                               // bFirstTime
    false,                                              // bInitialized
    131611U,                                            // fVersionInfo
    nullptr,                                            // fErrorFunction
    "EvaluationPath_GPU",                               // fFunctionName
    nullptr,                                            // fRTCallStack
    false,                                              // bDebugMode
    {3130694236U, 2616137409U, 972914731U, 129233577U}, // fSigWrd
    nullptr                                             // fSigMem
};

// End of code generation (EvaluationPath_GPU_data.cu)
