#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// EvaluationPath_GPU.cu
//
// Code generation for function 'EvaluationPath_GPU'
//

// Include files
#include "EvaluationPath_GPU.h"
#include "EvaluationPath_GPU_data.h"
#include "EvaluationPath_GPU_emxutil.h"
#include "EvaluationPath_GPU_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "rt_defines.h"
#include <cmath>
#include <cstring>

// Function Declarations
static __global__ void EvaluationPath_GPU_kernel1(const int32_T obj,
                                                  emxArray_real_T grade);

static __global__ void EvaluationPath_GPU_kernel10(const emxArray_real_T phaiR,
                                                   const real_T th,
                                                   const real_T r,
                                                   emxArray_real_T th_min);

static __global__ void EvaluationPath_GPU_kernel11(const real_T scale,
                                                   const real_T thg,
                                                   const int32_T i,
                                                   emxArray_real_T grade_temp);

static __global__ void
EvaluationPath_GPU_kernel12(const real_T scale,
                            const emxArray_real_T distance_obs, const int32_T i,
                            emxArray_boolean_T x);

static __global__ void EvaluationPath_GPU_kernel13(const int32_T i,
                                                   emxArray_real_T grade_temp);

static __global__ void
EvaluationPath_GPU_kernel14(const int32_T j, const int32_T i,
                            emxArray_boolean_T remove_sample,
                            emxArray_real_T grade_temp);

static __global__ void
EvaluationPath_GPU_kernel15(const emxArray_real_T grade_temp, const int32_T j,
                            const int32_T b_grade_temp,
                            const int32_T grade_dim0, emxArray_real_T grade);

static __global__ void EvaluationPath_GPU_kernel16(const emxArray_real_T grade,
                                                   const int32_T b_grade,
                                                   emxArray_real_T x);

static __global__ void EvaluationPath_GPU_kernel17(const int32_T vlen,
                                                   const emxArray_real_T x,
                                                   const int32_T npages,
                                                   const int32_T x_dim0,
                                                   emxArray_real_T FPMcost);

static __global__ void EvaluationPath_GPU_kernel18(const int32_T sz,
                                                   emxArray_real_T FPMcost);

static __global__ void EvaluationPath_GPU_kernel19(const real_T pu[42000],
                                                   real_T y[20000]);

static __global__ void
EvaluationPath_GPU_kernel2(const int32_T obj, emxArray_boolean_T remove_sample);

static __global__ void EvaluationPath_GPU_kernel20(const real_T y[20000],
                                                   real_T b_y[1000]);

static __global__ void EvaluationPath_GPU_kernel21(const struct0_T *obj,
                                                   const real_T pu[42000],
                                                   real_T a[20000]);

static __global__ void EvaluationPath_GPU_kernel22(const real_T a[20000],
                                                   real_T y[20000]);

static __global__ void EvaluationPath_GPU_kernel23(const real_T y[20000],
                                                   real_T b_y[1000]);

static __global__ void EvaluationPath_GPU_kernel24(const real_T pu[42000],
                                                   real_T a[20000]);

static __global__ void EvaluationPath_GPU_kernel25(const real_T a[20000],
                                                   real_T y[20000]);

static __global__ void EvaluationPath_GPU_kernel26(const real_T y[20000],
                                                   real_T dv[1000]);

static __global__ void EvaluationPath_GPU_kernel27(
    const struct0_T *obj, const real_T dv[1000], const real_T absxk,
    const real_T y[1000], const real_T scale, const real_T b_y[1000],
    const emxArray_real_T FPMcost, const int32_T b_iv, emxArray_real_T pw);

static __global__ void EvaluationPath_GPU_kernel3(const int32_T obj,
                                                  emxArray_real_T grade_temp);

static __global__ void
EvaluationPath_GPU_kernel4(const int32_T j, emxArray_boolean_T remove_sample);

static __global__ void
EvaluationPath_GPU_kernel5(const int32_T obj, emxArray_real_T th_min,
                           emxArray_real_T th_max, emxArray_real_T grade_obs,
                           emxArray_real_T a, emxArray_real_T phaiR,
                           emxArray_real_T PoQo, emxArray_real_T distance_obs);

static __global__ void EvaluationPath_GPU_kernel6(const real_T state[2],
                                                  const cell_wrap_0 preobs[22],
                                                  const int32_T i,
                                                  const int32_T npages,
                                                  emxArray_real_T distance_obs);

static __global__ void EvaluationPath_GPU_kernel7(const real_T state[2],
                                                  const cell_wrap_0 preobs[22],
                                                  const int32_T i, real_T x[2]);

static __global__ void EvaluationPath_GPU_kernel8(const real_T y,
                                                  const real_T scale,
                                                  const int32_T i,
                                                  emxArray_real_T PoQo);

static __global__ void
EvaluationPath_GPU_kernel9(const real_T r_obs, const real_T r_wheel,
                           const emxArray_real_T PoQo, const real_T ci,
                           const real_T x_re, emxArray_real_T a,
                           emxArray_real_T phaiR);

static void binary_expand_op(emxArray_real_T *pw,
                             const emxArray_real_T *FPMcost,
                             const real_T y[1000], const struct0_T *obj,
                             const real_T b_y[1000], const real_T dv[1000]);

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_boolean_T(emxArray_boolean_T *gpu,
                                           const emxArray_boolean_T *cpu);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static real_T rt_atan2d_snf(real_T u0, real_T u1);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel1(
    const int32_T obj, emxArray_real_T grade)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(obj);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    grade.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel10(
    const emxArray_real_T phaiR, const real_T th, const real_T r,
    emxArray_real_T th_min)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    th_min.data[0] =
        atan2(sin((r - th) + phaiR.data[0]), cos((r - th) + phaiR.data[0]));
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel11(
    const real_T scale, const real_T thg, const int32_T i,
    emxArray_real_T grade_temp)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    grade_temp.data[i] = fmin(
        -0.38197186342054879 * fabs(atan2(sin(thg), cos(thg))) + 1.0, scale);
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel12(
    const real_T scale, const emxArray_real_T distance_obs, const int32_T i,
    emxArray_boolean_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_i;
    b_i = static_cast<int32_T>(idx);
    x.data[b_i] = (distance_obs.data[b_i] < scale);
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel13(
    const int32_T i, emxArray_real_T grade_temp)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    grade_temp.data[i] = -100.0;
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel14(
    const int32_T j, const int32_T i, emxArray_boolean_T remove_sample,
    emxArray_real_T grade_temp)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    // 壁面の考慮
    grade_temp.data[i] = 0.0;
    remove_sample.data[j] = true;
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel15(
    const emxArray_real_T grade_temp, const int32_T j,
    const int32_T b_grade_temp, const int32_T grade_dim0, emxArray_real_T grade)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_grade_temp);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    grade.data[i + grade_dim0 * j] = grade_temp.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel16(
    const emxArray_real_T grade, const int32_T b_grade, emxArray_real_T x)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_grade);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = 1.0 - grade.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel17(
    const int32_T vlen, const emxArray_real_T x, const int32_T npages,
    const int32_T x_dim0, emxArray_real_T FPMcost)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(npages - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    int32_T xpageoffset;
    i = static_cast<int32_T>(idx);
    xpageoffset = i * x_dim0;
    FPMcost.data[i] = x.data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      FPMcost.data[i] += x.data[(xpageoffset + k) + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel18(
    const int32_T sz, emxArray_real_T FPMcost)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    FPMcost.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel19(
    const real_T pu[42000], real_T y[20000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 20000) {
    y[k] = pu[(((k % 20) << 1) + 42 * (k / 20)) + 1] *
           pu[(((k % 20) << 1) + 42 * (k / 20)) + 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel2(
    const int32_T obj, emxArray_boolean_T remove_sample)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(obj);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    remove_sample.data[i] = false;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel20(
    const real_T y[20000], real_T b_y[1000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 1000) {
    real_T t;
    int32_T xpageoffset;
    xpageoffset = i * 20;
    t = y[xpageoffset];
    for (int32_T k{0}; k < 19; k++) {
      t += y[(xpageoffset + k) + 1];
    }
    b_y[i] = t;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel21(
    const struct0_T *obj, const real_T pu[42000], real_T a[20000])
{
  uint64_T threadId;
  int32_T i;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId % 20ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(xpageoffset)) /
                           20ULL);
  if ((static_cast<int32_T>(i < 1000)) &&
      (static_cast<int32_T>(xpageoffset < 20))) {
    a[xpageoffset + 20 * i] = pu[(xpageoffset << 1) + 42 * i] - obj->Vref;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel22(
    const real_T a[20000], real_T y[20000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 20000) {
    real_T t;
    t = a[k];
    y[k] = t * t;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel23(
    const real_T y[20000], real_T b_y[1000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 1000) {
    real_T t;
    int32_T xpageoffset;
    xpageoffset = i * 20;
    t = y[xpageoffset];
    for (int32_T k{0}; k < 19; k++) {
      t += y[(xpageoffset + k) + 1];
    }
    b_y[i] = t;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel24(
    const real_T pu[42000], real_T a[20000])
{
  uint64_T threadId;
  int32_T i;
  int32_T xpageoffset;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  xpageoffset = static_cast<int32_T>(threadId % 20ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(xpageoffset)) /
                           20ULL);
  if ((static_cast<int32_T>(i < 1000)) &&
      (static_cast<int32_T>(xpageoffset < 20))) {
    a[xpageoffset + 20 * i] =
        pu[(xpageoffset << 1) + 42 * i] * pu[((xpageoffset << 1) + 42 * i) + 1];
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel25(
    const real_T a[20000], real_T y[20000])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 20000) {
    real_T t;
    t = a[k];
    y[k] = t * t;
  }
}

static __global__ __launch_bounds__(512, 1) void EvaluationPath_GPU_kernel26(
    const real_T y[20000], real_T dv[1000])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 1000) {
    real_T t;
    int32_T xpageoffset;
    xpageoffset = i * 20;
    t = y[xpageoffset];
    for (int32_T k{0}; k < 19; k++) {
      t += y[(xpageoffset + k) + 1];
    }
    dv[i] = t;
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel27(
    const struct0_T *obj, const real_T dv[1000], const real_T absxk,
    const real_T y[1000], const real_T scale, const real_T b_y[1000],
    const emxArray_real_T FPMcost, const int32_T b_iv, emxArray_real_T pw)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_iv);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    pw.data[i] =
        (FPMcost.data[i] + (b_y[i] * scale + y[i] * absxk)) + dv[i] * obj->W;
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel3(
    const int32_T obj, emxArray_real_T grade_temp)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(obj);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    grade_temp.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel4(
    const int32_T j, emxArray_boolean_T remove_sample)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    remove_sample.data[j] = false;
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel5(
    const int32_T obj, emxArray_real_T th_min, emxArray_real_T th_max,
    emxArray_real_T grade_obs, emxArray_real_T a, emxArray_real_T phaiR,
    emxArray_real_T PoQo, emxArray_real_T distance_obs)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(obj);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    distance_obs.data[i] = 0.0;
    PoQo.data[i] = 0.0;
    phaiR.data[i] = 0.0;
    a.data[i] = 0.0;
    grade_obs.data[i] = 1.0;
    th_max.data[i] = 0.0;
    th_min.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void EvaluationPath_GPU_kernel6(
    const real_T state[2], const cell_wrap_0 preobs[22], const int32_T i,
    const int32_T npages, emxArray_real_T distance_obs)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(npages);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    real_T absxk;
    real_T scale;
    real_T t;
    real_T y;
    int32_T b_i;
    b_i = static_cast<int32_T>(idx);
    scale = 3.3121686421112381E-170;
    absxk = fabs(preobs[i + 1].f1[0] - state[0]);
    if (absxk > 3.3121686421112381E-170) {
      y = 1.0;
      scale = absxk;
    } else {
      t = absxk / 3.3121686421112381E-170;
      y = t * t;
    }
    absxk = fabs(preobs[i + 1].f1[1] - state[1]);
    if (absxk > scale) {
      t = scale / absxk;
      y = y * t * t + 1.0;
      scale = absxk;
    } else {
      t = absxk / scale;
      y += t * t;
    }
    distance_obs.data[b_i] = scale * sqrt(y);
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel7(
    const real_T state[2], const cell_wrap_0 preobs[22], const int32_T i,
    real_T x[2])
{
  uint64_T threadId;
  int32_T b_i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_i = static_cast<int32_T>(threadId);
  if (b_i < 2) {
    //          tho(i,1) = atan2((obs{i,1}(1,2) - y),(obs{i,1}(1,1) - x)) - th;
    //          PoQo(i,1) = norm(obs{i,1} - [x y]);
    x[b_i] = preobs[i + 2].f1[b_i] - state[b_i];
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel8(
    const real_T y, const real_T scale, const int32_T i, emxArray_real_T PoQo)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    PoQo.data[i] = scale * sqrt(y);
  }
}

static __global__ __launch_bounds__(32, 1) void EvaluationPath_GPU_kernel9(
    const real_T r_obs, const real_T r_wheel, const emxArray_real_T PoQo,
    const real_T ci, const real_T x_re, emxArray_real_T a,
    emxArray_real_T phaiR)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    phaiR.data[0] = hypot(x_re, ci);
    a.data[0] = (2.5 - PoQo.data[0]) / (2.5 - (r_wheel + r_obs));
  }
}

static void binary_expand_op(emxArray_real_T *pw,
                             const emxArray_real_T *FPMcost,
                             const real_T y[1000], const struct0_T *obj,
                             const real_T b_y[1000], const real_T dv[1000])
{
  int32_T aux_0_1;
  int32_T aux_1_1;
  int32_T aux_2_1;
  int32_T b_unnamed_idx_1;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  int32_T stride_2_1;
  int32_T unnamed_idx_1;
  unnamed_idx_1 = static_cast<int32_T>(obj->NP);
  b_unnamed_idx_1 = static_cast<int32_T>(obj->NP);
  i = pw->size[0] * pw->size[1];
  pw->size[0] = 1;
  if (static_cast<int32_T>(obj->NP) == 1) {
    pw->size[1] = FPMcost->size[1];
  } else {
    pw->size[1] = static_cast<int32_T>(obj->NP);
  }
  emxEnsureCapacity_real_T(pw, i);
  stride_0_1 = (FPMcost->size[1] != 1);
  stride_1_1 = (static_cast<int32_T>(obj->NP) != 1);
  stride_2_1 = (static_cast<int32_T>(obj->NP) != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  aux_2_1 = 0;
  if (b_unnamed_idx_1 == 1) {
    if (unnamed_idx_1 == 1) {
      b_unnamed_idx_1 = FPMcost->size[1];
    } else {
      b_unnamed_idx_1 = unnamed_idx_1;
    }
  }
  for (i = 0; i < b_unnamed_idx_1; i++) {
    pw->data[i] = (FPMcost->data[aux_0_1] +
                   (y[aux_1_1] * obj->R[1] + b_y[aux_1_1] * obj->R[0])) +
                  dv[aux_2_1] * obj->W;
    aux_2_1 += stride_2_1;
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
}

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu)
{
  boolean_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(boolean_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(boolean_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(boolean_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  real_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(real_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(real_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_boolean_T(emxArray_boolean_T *gpu,
                                           const emxArray_boolean_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(boolean_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(boolean_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      hipFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    hipMalloc(&gpu->data, gpu->allocatedSize * sizeof(real_T));
  }
  hipMemcpy(gpu->data, cpu->data, actualSize * sizeof(real_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(boolean_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data, actualSize * sizeof(real_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_boolean_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

static real_T rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else if (std::isinf(u0) && std::isinf(u1)) {
    int32_T b;
    int32_T c;
    if (u1 > 0.0) {
      b = 1;
    } else {
      b = -1;
    }
    if (u0 > 0.0) {
      c = 1;
    } else {
      c = -1;
    }
    y = std::atan2(static_cast<real_T>(c), static_cast<real_T>(b));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = std::atan2(u0, u1);
  }
  return y;
}

void EvaluationPath_GPU(const struct0_T *obj, const real_T px[63000],
                        const real_T pu[42000], const cell_wrap_0 preobs[22],
                        emxArray_real_T *pw, emxArray_boolean_T *remove_sample)
{
  cell_wrap_0(*gpu_preobs)[22];
  dim3 block;
  dim3 grid;
  emxArray_boolean_T c_gpu_x;
  emxArray_boolean_T gpu_remove_sample;
  emxArray_boolean_T *x;
  emxArray_real_T b_gpu_a;
  emxArray_real_T gpu_FPMcost;
  emxArray_real_T gpu_PoQo;
  emxArray_real_T gpu_distance_obs;
  emxArray_real_T gpu_grade;
  emxArray_real_T gpu_grade_obs;
  emxArray_real_T gpu_grade_temp;
  emxArray_real_T gpu_phaiR;
  emxArray_real_T gpu_pw;
  emxArray_real_T gpu_th_max;
  emxArray_real_T gpu_th_min;
  emxArray_real_T gpu_x;
  emxArray_real_T *FPMcost;
  emxArray_real_T *PoQo;
  emxArray_real_T *a;
  emxArray_real_T *b_x;
  emxArray_real_T *distance_obs;
  emxArray_real_T *grade;
  emxArray_real_T *grade_obs;
  emxArray_real_T *grade_temp;
  emxArray_real_T *phaiR;
  emxArray_real_T *th_max;
  emxArray_real_T *th_min;
  struct0_T *gpu_obj;
  real_T(*gpu_pu)[42000];
  real_T(*gpu_a)[20000];
  real_T(*gpu_y)[20000];
  real_T b_y[1000];
  real_T dv[1000];
  real_T y[1000];
  real_T(*b_gpu_y)[1000];
  real_T(*c_gpu_y)[1000];
  real_T(*gpu_dv)[1000];
  real_T c_x[2];
  real_T state[2];
  real_T(*b_gpu_x)[2];
  real_T(*gpu_state)[2];
  real_T absxk;
  real_T scale;
  int32_T b_iv[2];
  int32_T b_i;
  int32_T b_obj;
  int32_T i;
  int32_T npages;
  int32_T vlen;
  uint32_T sz[2];
  boolean_T FPMcost_dirtyOnGpu;
  boolean_T PoQo_dirtyOnGpu;
  boolean_T a_dirtyOnGpu;
  boolean_T grade_obs_dirtyOnCpu;
  boolean_T grade_obs_dirtyOnGpu;
  boolean_T phaiR_dirtyOnGpu;
  boolean_T preobs_dirtyOnCpu;
  boolean_T pw_dirtyOnGpu;
  boolean_T remove_sample_dirtyOnCpu;
  boolean_T remove_sample_dirtyOnGpu;
  boolean_T th_max_dirtyOnCpu;
  boolean_T th_max_dirtyOnGpu;
  boolean_T th_min_dirtyOnGpu;
  boolean_T x_dirtyOnGpu;
  boolean_T xneg;
  gpuEmxReset_boolean_T(&c_gpu_x);
  hipMalloc(&b_gpu_x, 16ULL);
  hipMalloc(&gpu_preobs, 352ULL);
  hipMalloc(&gpu_state, 16ULL);
  gpuEmxReset_real_T(&gpu_pw);
  gpuEmxReset_real_T(&gpu_distance_obs);
  gpuEmxReset_real_T(&gpu_PoQo);
  gpuEmxReset_real_T(&gpu_phaiR);
  gpuEmxReset_real_T(&b_gpu_a);
  gpuEmxReset_real_T(&gpu_grade_obs);
  gpuEmxReset_real_T(&gpu_th_max);
  gpuEmxReset_real_T(&gpu_th_min);
  hipMalloc(&gpu_dv, 8000ULL);
  hipMalloc(&c_gpu_y, 8000ULL);
  hipMalloc(&gpu_a, 160000ULL);
  hipMalloc(&gpu_obj, 1008072ULL);
  hipMalloc(&b_gpu_y, 8000ULL);
  hipMalloc(&gpu_y, 160000ULL);
  hipMalloc(&gpu_pu, 336000ULL);
  gpuEmxReset_real_T(&gpu_FPMcost);
  gpuEmxReset_real_T(&gpu_x);
  gpuEmxReset_real_T(&gpu_grade_temp);
  gpuEmxReset_boolean_T(&gpu_remove_sample);
  gpuEmxReset_real_T(&gpu_grade);
  x_dirtyOnGpu = false;
  PoQo_dirtyOnGpu = false;
  phaiR_dirtyOnGpu = false;
  a_dirtyOnGpu = false;
  grade_obs_dirtyOnGpu = false;
  th_max_dirtyOnGpu = false;
  th_min_dirtyOnGpu = false;
  FPMcost_dirtyOnGpu = false;
  grade_obs_dirtyOnCpu = false;
  th_max_dirtyOnCpu = false;
  preobs_dirtyOnCpu = true;
  pw_dirtyOnGpu = false;
  remove_sample_dirtyOnGpu = false;
  remove_sample_dirtyOnCpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&grade, 2, true);
  // EVALUATIONPATH この関数の概要をここに記述
  //    詳細説明をここに記述
  i = grade->size[0] * grade->size[1];
  grade->size[0] = static_cast<int32_T>(obj->K - 1.0);
  grade->size[1] = static_cast<int32_T>(obj->NP);
  emxEnsureCapacity_real_T(grade, i);
  gpuEmxEnsureCapacity_real_T(grade, &gpu_grade);
  b_obj =
      static_cast<int32_T>(obj->K - 1.0) * static_cast<int32_T>(obj->NP) - 1;
  xneg = mwGetLaunchParameters1D(static_cast<real_T>(b_obj + 1LL), &grid,
                                 &block, 1024U, 65535U);
  if (xneg) {
    EvaluationPath_GPU_kernel1<<<grid, block>>>(b_obj, gpu_grade);
  }
  i = remove_sample->size[0] * remove_sample->size[1];
  remove_sample->size[0] = 1;
  remove_sample->size[1] = static_cast<int32_T>(obj->NP);
  emxEnsureCapacity_boolean_T(remove_sample, i);
  xneg = mwGetLaunchParameters1D(
      static_cast<real_T>((static_cast<int32_T>(obj->NP) - 1) + 1LL), &grid,
      &block, 1024U, 65535U);
  if (xneg) {
    gpuEmxMemcpyCpuToGpu_boolean_T(&gpu_remove_sample, remove_sample);
    EvaluationPath_GPU_kernel2<<<grid, block>>>(
        static_cast<int32_T>(obj->NP) - 1, gpu_remove_sample);
    remove_sample_dirtyOnCpu = false;
    remove_sample_dirtyOnGpu = true;
  }
  //      ticBytes(gcp);
  b_i = static_cast<int32_T>(obj->NP);
  emxInit_real_T(&grade_temp, 1, true);
  emxInit_real_T(&distance_obs, 1, true);
  emxInit_real_T(&PoQo, 1, true);
  emxInit_real_T(&phaiR, 1, true);
  emxInit_real_T(&a, 1, true);
  emxInit_real_T(&grade_obs, 1, true);
  emxInit_real_T(&th_max, 1, true);
  emxInit_real_T(&th_min, 1, true);
  emxInit_boolean_T(&x, 1, true);
  for (int32_T j{0}; j < b_i; j++) {
    i = grade_temp->size[0];
    grade_temp->size[0] = static_cast<int32_T>(obj->K - 1.0);
    emxEnsureCapacity_real_T(grade_temp, i);
    gpuEmxEnsureCapacity_real_T(grade_temp, &gpu_grade_temp);
    xneg = mwGetLaunchParameters1D(
        static_cast<real_T>((static_cast<int32_T>(obj->K - 1.0) - 1) + 1LL),
        &grid, &block, 1024U, 65535U);
    if (xneg) {
      EvaluationPath_GPU_kernel3<<<grid, block>>>(
          static_cast<int32_T>(obj->K - 1.0) - 1, gpu_grade_temp);
    }
    if (remove_sample_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_boolean_T(&gpu_remove_sample, remove_sample);
    }
    EvaluationPath_GPU_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        j, gpu_remove_sample);
    remove_sample_dirtyOnCpu = false;
    remove_sample_dirtyOnGpu = true;
    b_obj = static_cast<int32_T>(obj->K - 1.0);
    for (int32_T c_i{0}; c_i < b_obj; c_i++) {
      real_T r;
      real_T r_obs;
      real_T r_wheel;
      real_T th;
      real_T thg;
      boolean_T exitg1;
      boolean_T state_dirtyOnCpu;
      //  			[~,grade(i,j)] =
      //  FPM_2_6(obj,px(1,i+1,j),px(2,i+1,j),px(3,i+1,j),preobs{i+2,1},preobs{i+1,1});
      th = px[(3 * (c_i + 1) + 63 * j) + 2];
      // goal = obj.goal;
      r_wheel = obj->r_wheel;
      r_obs = obj->r_obs;
      //     eps = 1.0;
      //      Q = [1:-0.1:0.1];
      i = distance_obs->size[0];
      distance_obs->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(distance_obs, i);
      gpuEmxEnsureCapacity_real_T(distance_obs, &gpu_distance_obs);
      i = PoQo->size[0];
      PoQo->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(PoQo, i);
      gpuEmxEnsureCapacity_real_T(PoQo, &gpu_PoQo);
      i = phaiR->size[0];
      phaiR->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(phaiR, i);
      gpuEmxEnsureCapacity_real_T(phaiR, &gpu_phaiR);
      i = a->size[0];
      a->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(a, i);
      gpuEmxEnsureCapacity_real_T(a, &b_gpu_a);
      i = grade_obs->size[0];
      grade_obs->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(grade_obs, i);
      if (!grade_obs_dirtyOnCpu) {
        gpuEmxEnsureCapacity_real_T(grade_obs, &gpu_grade_obs);
      }
      i = th_max->size[0];
      th_max->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(th_max, i);
      if (!th_max_dirtyOnCpu) {
        gpuEmxEnsureCapacity_real_T(th_max, &gpu_th_max);
      }
      i = th_min->size[0];
      th_min->size[0] = static_cast<int32_T>(obj->ObsNum);
      emxEnsureCapacity_real_T(th_min, i);
      gpuEmxEnsureCapacity_real_T(th_min, &gpu_th_min);
      xneg = mwGetLaunchParameters1D(
          static_cast<real_T>((static_cast<int32_T>(obj->ObsNum) - 1) + 1LL),
          &grid, &block, 1024U, 65535U);
      if (xneg) {
        if (th_max_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_th_max, th_max);
        }
        if (grade_obs_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_grade_obs, grade_obs);
        }
        EvaluationPath_GPU_kernel5<<<grid, block>>>(
            static_cast<int32_T>(obj->ObsNum) - 1, gpu_th_min, gpu_th_max,
            gpu_grade_obs, b_gpu_a, gpu_phaiR, gpu_PoQo, gpu_distance_obs);
        a_dirtyOnGpu = true;
        PoQo_dirtyOnGpu = true;
        th_min_dirtyOnGpu = true;
        phaiR_dirtyOnGpu = true;
        grade_obs_dirtyOnCpu = false;
        grade_obs_dirtyOnGpu = true;
        th_max_dirtyOnCpu = false;
        th_max_dirtyOnGpu = true;
      }
      //      OBS_NOW = cell2mat(obs_now);
      //      OBS = cell2mat(obs);
      state[0] = px[3 * (c_i + 1) + 63 * j];
      state[1] = px[(3 * (c_i + 1) + 63 * j) + 1];
      state_dirtyOnCpu = true;
      //  PMFのプロットする際にコメント外す
      //      resolution = 0.1;%PMFの分解能[deg]
      //      GRADE_TRGT = zeros(1,360/resolution);
      //      GRADE_OBS = ones(1,360/resolution);
      //      GRADE_obs = ones(obsnum,360/resolution);
      //      GRADE_ANGL = [-pi + resolution*pi/180:resolution*pi/180:pi];
      //      distance = norm(goal-[x y]);
      //      tic
      //      distance_obs = sqrt(sum((OBS_NOW - state).^2,2));
      //      distance_obs = vecnorm(OBS_NOW-state,2,2);
      //      toc
      //      tic
      xneg = mwGetLaunchParameters1D(
          static_cast<real_T>((static_cast<int32_T>(obj->ObsNum) - 1) + 1LL),
          &grid, &block, 1024U, 65535U);
      if (xneg) {
        hipMemcpy(*gpu_state, state, 16ULL, hipMemcpyHostToDevice);
        state_dirtyOnCpu = false;
        if (preobs_dirtyOnCpu) {
          hipMemcpy(*gpu_preobs, preobs, 352ULL, hipMemcpyHostToDevice);
        }
        preobs_dirtyOnCpu = false;
        EvaluationPath_GPU_kernel6<<<grid, block>>>(
            *gpu_state, *gpu_preobs, c_i, static_cast<int32_T>(obj->ObsNum) - 1,
            gpu_distance_obs);
      }
      //      toc
      //      tic
      //      for i = 1:obsnum
      //          distance_obs(i,1) = norm(obs_now{i,1} - [x y]);
      //      end
      //     toc
      //      if distance < eps
      //          g_a = distance / eps;
      //          g_b = eta * g_a;
      //      end
      scale = px[(3 * (c_i + 1) + 63 * j) + 1];
      absxk = px[3 * (c_i + 1) + 63 * j];
      thg = rt_atan2d_snf(obj->target[(((c_i + 1) << 1) + 42 * j) + 1] - scale,
                          obj->target[((c_i + 1) << 1) + 42 * j] - absxk) -
            px[(3 * (c_i + 1) + 63 * j) + 2];
      r = rt_atan2d_snf(preobs[c_i + 2].f1[1] - scale,
                        preobs[c_i + 2].f1[0] - absxk);
      //      PoQo = sqrt(sum((OBS - state).^2,2));
      //      PoQo = vecnorm(OBS-state,2,2);
      //      phaiR = abs(asin((r_wheel + r_obs)./PoQo));
      //      a = (alpha - PoQo)/(alpha - (r_wheel + r_obs));
      npages = static_cast<int32_T>(obj->ObsNum);
      for (i = 0; i < npages; i++) {
        real_T c_y;
        real_T ci;
        real_T t;
        real_T x_re;
        //          tho(i,1) = atan2((obs{i,1}(1,2) - y),(obs{i,1}(1,1) - x)) -
        //          th; PoQo(i,1) = norm(obs{i,1} - [x y]);
        if (state_dirtyOnCpu) {
          hipMemcpy(*gpu_state, state, 16ULL, hipMemcpyHostToDevice);
        }
        state_dirtyOnCpu = false;
        if (preobs_dirtyOnCpu) {
          hipMemcpy(*gpu_preobs, preobs, 352ULL, hipMemcpyHostToDevice);
        }
        preobs_dirtyOnCpu = false;
        EvaluationPath_GPU_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            *gpu_state, *gpu_preobs, c_i, *b_gpu_x);
        scale = 3.3121686421112381E-170;
        hipMemcpy(c_x, *b_gpu_x, 16ULL, hipMemcpyDeviceToHost);
        absxk = std::abs(c_x[0]);
        if (absxk > 3.3121686421112381E-170) {
          c_y = 1.0;
          scale = absxk;
        } else {
          t = absxk / 3.3121686421112381E-170;
          c_y = t * t;
        }
        absxk = std::abs(c_x[1]);
        if (absxk > scale) {
          t = scale / absxk;
          c_y = c_y * t * t + 1.0;
          scale = absxk;
        } else {
          t = absxk / scale;
          c_y += t * t;
        }
        EvaluationPath_GPU_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            c_y, scale, i, gpu_PoQo);
        gpuEmxMemcpyGpuToCpu_real_T(PoQo, &gpu_PoQo);
        PoQo_dirtyOnGpu = false;
        x_re = (r_wheel + r_obs) / PoQo->data[0];
        if (!(std::abs(x_re) > 1.0)) {
          x_re = std::asin(x_re);
          ci = 0.0;
        } else {
          real_T b_yi;
          real_T b_yr;
          real_T sbi;
          real_T scaleB;
          real_T tmp;
          real_T yi;
          real_T yr;
          if (x_re + 1.0 < 0.0) {
            yr = 0.0;
            yi = std::sqrt(-(x_re + 1.0));
          } else {
            yr = std::sqrt(x_re + 1.0);
            yi = 0.0;
          }
          if (1.0 - x_re < 0.0) {
            b_yr = 0.0;
            b_yi = std::sqrt(-(1.0 - x_re));
          } else {
            b_yr = std::sqrt(1.0 - x_re);
            b_yi = 0.0;
          }
          if ((b_yi == 0.0) && (yi == 0.0)) {
            tmp = b_yr * yr;
          } else {
            tmp = b_yr * yr - b_yi * yi;
            if ((std::isinf(tmp) || std::isnan(tmp)) && (!std::isnan(b_yr)) &&
                (!std::isnan(yr))) {
              if (b_yr > b_yi) {
                absxk = 1.0;
                c_y = b_yi / b_yr;
                scale = b_yr;
              } else if (b_yi > b_yr) {
                absxk = b_yr / b_yi;
                c_y = 1.0;
                scale = b_yi;
              } else {
                absxk = 1.0;
                c_y = 1.0;
                scale = b_yr;
              }
              if (yr > yi) {
                t = 1.0;
                sbi = yi / yr;
                scaleB = yr;
              } else if (yi > yr) {
                t = yr / yi;
                sbi = 1.0;
                scaleB = yi;
              } else {
                t = 1.0;
                sbi = 1.0;
                scaleB = yr;
              }
              if ((!std::isinf(scale)) && (!std::isnan(scale)) &&
                  ((!std::isinf(scaleB)) && (!std::isnan(scaleB)))) {
                xneg = true;
              } else {
                xneg = false;
              }
              if (std::isnan(tmp) || (std::isinf(tmp) && xneg)) {
                tmp = absxk * t - c_y * sbi;
                if (tmp != 0.0) {
                  tmp = tmp * scale * scaleB;
                } else if ((std::isinf(scale) &&
                            ((yr == 0.0) || (yi == 0.0))) ||
                           (std::isinf(scaleB) &&
                            ((b_yr == 0.0) || (b_yi == 0.0)))) {
                  scale = b_yr * yr;
                  absxk = b_yi * yi;
                  if (std::isnan(scale)) {
                    scale = 0.0;
                  }
                  if (std::isnan(absxk)) {
                    absxk = 0.0;
                  }
                  tmp = scale - absxk;
                }
              }
            }
          }
          if ((-b_yi == 0.0) && (yi == 0.0)) {
            ci = 0.0;
          } else {
            ci = b_yr * yi + -b_yi * yr;
            if ((std::isinf(ci) || std::isnan(ci)) && (!std::isnan(b_yr)) &&
                (!std::isnan(yr))) {
              scale = std::abs(-b_yi);
              if (b_yr > scale) {
                absxk = 1.0;
                c_y = -b_yi / b_yr;
                scale = b_yr;
              } else if (scale > b_yr) {
                absxk = b_yr / scale;
                if (-b_yi < 0.0) {
                  c_y = -1.0;
                } else {
                  c_y = 1.0;
                }
              } else {
                absxk = 1.0;
                if (-b_yi < 0.0) {
                  c_y = -1.0;
                } else {
                  c_y = 1.0;
                }
                scale = b_yr;
              }
              if (yr > yi) {
                t = 1.0;
                sbi = yi / yr;
                scaleB = yr;
              } else if (yi > yr) {
                t = yr / yi;
                sbi = 1.0;
                scaleB = yi;
              } else {
                t = 1.0;
                sbi = 1.0;
                scaleB = yr;
              }
              if ((!std::isinf(scale)) && (!std::isnan(scale)) &&
                  ((!std::isinf(scaleB)) && (!std::isnan(scaleB)))) {
                xneg = true;
              } else {
                xneg = false;
              }
              if (std::isnan(ci) || (std::isinf(ci) && xneg)) {
                ci = absxk * sbi + c_y * t;
                if (ci != 0.0) {
                  ci = ci * scale * scaleB;
                } else if ((std::isinf(scale) &&
                            ((yr == 0.0) || (yi == 0.0))) ||
                           (std::isinf(scaleB) &&
                            ((b_yr == 0.0) || (-b_yi == 0.0)))) {
                  absxk = b_yr * yi;
                  scale = -b_yi * yr;
                  if (std::isnan(absxk)) {
                    absxk = 0.0;
                  }
                  if (std::isnan(scale)) {
                    scale = 0.0;
                  }
                  ci = absxk + scale;
                }
              }
            }
          }
          xneg = (ci < 0.0);
          if (xneg) {
            ci = -ci;
          }
          if (ci >= 2.68435456E+8) {
            ci = std::log(ci) + 0.69314718055994529;
          } else if (ci > 2.0) {
            ci = std::log(2.0 * ci + 1.0 / (std::sqrt(ci * ci + 1.0) + ci));
          } else {
            t = ci * ci;
            ci += t / (std::sqrt(t + 1.0) + 1.0);
            scale = std::abs(ci);
            if ((scale > 4.503599627370496E+15) ||
                (std::isinf(ci) || std::isnan(ci))) {
              ci++;
              ci = std::log(ci);
            } else if (!(scale < 2.2204460492503131E-16)) {
              ci = std::log(ci + 1.0) * (ci / ((ci + 1.0) - 1.0));
            }
          }
          if (xneg) {
            ci = -ci;
          }
          x_re = rt_atan2d_snf(x_re, tmp);
        }
        EvaluationPath_GPU_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            r_obs, r_wheel, gpu_PoQo, ci, x_re, b_gpu_a, gpu_phaiR);
        a_dirtyOnGpu = true;
        phaiR_dirtyOnGpu = true;
      }
      //      grade_trgt = ((g_a-g_b)/pi)*(pi-abs(thg))+g_b;
      npages = static_cast<int32_T>(obj->ObsNum);
      if (0 <= static_cast<int32_T>(obj->ObsNum) - 1) {
        EvaluationPath_GPU_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            gpu_phaiR, th, r, gpu_th_min);
        th_min_dirtyOnGpu = true;
      }
      for (i = 0; i < npages; i++) {
        //          grade_obs(i,1) = 1.0;
        //          if tho(i,1) - phaiR(i,1) <= 0.0 && 0.0 <= tho(i,1) +
        //          phaiR(i,1)
        if (th_max_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(th_max, &gpu_th_max);
        }
        if (phaiR_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_real_T(phaiR, &gpu_phaiR);
        }
        phaiR_dirtyOnGpu = false;
        th_max->data[i] = rt_atan2d_snf(std::sin((r - th) - phaiR->data[0]),
                                        std::cos((r - th) - phaiR->data[0]));
        th_max_dirtyOnGpu = false;
        th_max_dirtyOnCpu = true;
        //          th_max_local(i,1) = tho(i,1) - phaiR(i,1);
        //          th_min_local(i,1) = tho(i,1) + phaiR(i,1);
        if (th_max->data[0] <= 0.0) {
          if (th_min_dirtyOnGpu) {
            gpuEmxMemcpyGpuToCpu_real_T(th_min, &gpu_th_min);
          }
          th_min_dirtyOnGpu = false;
          if (0.0 <= th_min->data[0]) {
            if (PoQo_dirtyOnGpu) {
              gpuEmxMemcpyGpuToCpu_real_T(PoQo, &gpu_PoQo);
            }
            PoQo_dirtyOnGpu = false;
            if (PoQo->data[0] < 2.5) {
              if (grade_obs_dirtyOnGpu) {
                gpuEmxMemcpyGpuToCpu_real_T(grade_obs, &gpu_grade_obs);
              }
              if (a_dirtyOnGpu) {
                gpuEmxMemcpyGpuToCpu_real_T(a, &b_gpu_a);
              }
              a_dirtyOnGpu = false;
              grade_obs->data[0] = 1.0 - a->data[0];
              grade_obs_dirtyOnGpu = false;
              grade_obs_dirtyOnCpu = true;
              //
            } else {
              if (grade_obs_dirtyOnGpu) {
                gpuEmxMemcpyGpuToCpu_real_T(grade_obs, &gpu_grade_obs);
              }
              grade_obs->data[0] = 1.0;
              grade_obs_dirtyOnGpu = false;
              grade_obs_dirtyOnCpu = true;
            }
          }
        }
      }
      vlen = grade_obs->size[0];
      if (grade_obs_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_real_T(grade_obs, &gpu_grade_obs);
      }
      grade_obs_dirtyOnGpu = false;
      scale = grade_obs->data[0];
      for (i = 0; i <= vlen - 2; i++) {
        if (std::isnan(grade_obs->data[i + 1])) {
          xneg = false;
        } else if (std::isnan(scale)) {
          xneg = true;
        } else {
          xneg = (scale > grade_obs->data[i + 1]);
        }
        if (xneg) {
          scale = grade_obs->data[i + 1];
        }
      }
      EvaluationPath_GPU_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          scale, thg, c_i, gpu_grade_temp);
      //      phai = -pi + resolution*pi/180;
      //      for j = 1:360/resolution
      //          phaierror = atan2(sin(thg - phai),cos(thg - phai));
      //          GRADE_TRGT(1,j) = -((g_a - g_b)/pi)*abs(phaierror) + g_a;
      //          for i = 1:obsnum
      //              if th_min_local(i,1) > pi || th_max_local(i,1) < -pi
      //                  if or(th_max(i,1) <= phai && phai <= pi,th_min(i,1) >=
      //                  phai && phai >= -pi)
      //                      if PoQo(i,1) < alpha
      //                          GRADE_obs(i,j) = 1 - a(i,1);
      //                      end
      //                  end
      //              else
      //                  if th_max(i,1) <= phai && phai <= th_min(i,1)
      //                      if PoQo(i,1) < alpha
      //                          GRADE_obs(i,j) = 1 - a(i,1);
      //                      end
      //                  end
      //              end
      //          end
      //          phai = phai + resolution*pi/180;
      //      end
      //      GRADE_OBS = min(GRADE_obs,[],1);
      //      Grade = max(min([GRADE_TRGT;GRADE_OBS],[],1));
      i = distance_obs->size[0] - 1;
      npages = x->size[0];
      x->size[0] = distance_obs->size[0];
      emxEnsureCapacity_boolean_T(x, npages);
      gpuEmxEnsureCapacity_boolean_T(x, &c_gpu_x);
      xneg = mwGetLaunchParameters1D(static_cast<real_T>(i + 1LL), &grid,
                                     &block, 1024U, 65535U);
      if (xneg) {
        EvaluationPath_GPU_kernel12<<<grid, block>>>(
            obj->r_wheel + obj->r_obs, gpu_distance_obs, i, c_gpu_x);
        x_dirtyOnGpu = true;
      }
      xneg = false;
      npages = 1;
      exitg1 = false;
      while ((!exitg1) && (npages <= x->size[0])) {
        if (x_dirtyOnGpu) {
          gpuEmxMemcpyGpuToCpu_boolean_T(x, &c_gpu_x);
        }
        x_dirtyOnGpu = false;
        if (x->data[npages - 1]) {
          xneg = true;
          exitg1 = true;
        } else {
          npages++;
        }
      }
      if (xneg) {
        EvaluationPath_GPU_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            c_i, gpu_grade_temp);
      }
      scale = px[3 * (c_i + 1) + 63 * j];
      if ((scale < obj->limit[((c_i + 1) << 2) + 84 * j]) ||
          (scale > obj->limit[(((c_i + 1) << 2) + 84 * j) + 1]) ||
          (px[(3 * (c_i + 1) + 63 * j) + 1] <
           obj->limit[(((c_i + 1) << 2) + 84 * j) + 2]) ||
          (px[(3 * (c_i + 1) + 63 * j) + 1] >
           obj->limit[(((c_i + 1) << 2) + 84 * j) + 3])) {
        // 壁面の考慮
        EvaluationPath_GPU_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            j, c_i, gpu_remove_sample, gpu_grade_temp);
      }
    }
    b_obj = grade_temp->size[0] - 1;
    npages = grade->size[0];
    xneg = mwGetLaunchParameters1D(static_cast<real_T>(b_obj + 1LL), &grid,
                                   &block, 1024U, 65535U);
    if (xneg) {
      EvaluationPath_GPU_kernel15<<<grid, block>>>(gpu_grade_temp, j, b_obj,
                                                   npages, gpu_grade);
    }
  }
  emxFree_boolean_T(&x);
  emxFree_real_T(&th_min);
  emxFree_real_T(&th_max);
  emxFree_real_T(&grade_obs);
  emxFree_real_T(&a);
  emxFree_real_T(&phaiR);
  emxFree_real_T(&PoQo);
  emxFree_real_T(&distance_obs);
  emxFree_real_T(&grade_temp);
  emxInit_real_T(&b_x, 2, true);
  //      tocBytes(gcp);
  i = b_x->size[0] * b_x->size[1];
  b_x->size[0] = grade->size[0];
  b_x->size[1] = grade->size[1];
  emxEnsureCapacity_real_T(b_x, i);
  gpuEmxEnsureCapacity_real_T(b_x, &gpu_x);
  b_obj = grade->size[0] * grade->size[1] - 1;
  xneg = mwGetLaunchParameters1D(static_cast<real_T>(b_obj + 1LL), &grid,
                                 &block, 1024U, 65535U);
  if (xneg) {
    EvaluationPath_GPU_kernel16<<<grid, block>>>(gpu_grade, b_obj, gpu_x);
  }
  emxFree_real_T(&grade);
  vlen = b_x->size[0];
  emxInit_real_T(&FPMcost, 2, true);
  if ((b_x->size[0] == 0) || (b_x->size[1] == 0)) {
    for (i = 0; i < 2; i++) {
      sz[i] = static_cast<uint32_T>(b_x->size[i]);
    }
    i = FPMcost->size[0] * FPMcost->size[1];
    FPMcost->size[0] = 1;
    FPMcost->size[1] = static_cast<int32_T>(sz[1]);
    emxEnsureCapacity_real_T(FPMcost, i);
    gpuEmxEnsureCapacity_real_T(FPMcost, &gpu_FPMcost);
    xneg = mwGetLaunchParameters1D(
        static_cast<real_T>((static_cast<int32_T>(sz[1]) - 1) + 1LL), &grid,
        &block, 1024U, 65535U);
    if (xneg) {
      EvaluationPath_GPU_kernel18<<<grid, block>>>(
          static_cast<int32_T>(sz[1]) - 1, gpu_FPMcost);
      FPMcost_dirtyOnGpu = true;
    }
  } else {
    npages = b_x->size[1];
    i = FPMcost->size[0] * FPMcost->size[1];
    FPMcost->size[0] = 1;
    FPMcost->size[1] = b_x->size[1];
    emxEnsureCapacity_real_T(FPMcost, i);
    gpuEmxEnsureCapacity_real_T(FPMcost, &gpu_FPMcost);
    b_obj = b_x->size[0];
    xneg = mwGetLaunchParameters1D(static_cast<real_T>((npages - 1) + 1LL),
                                   &grid, &block, 1024U, 65535U);
    if (xneg) {
      EvaluationPath_GPU_kernel17<<<grid, block>>>(vlen, gpu_x, npages, b_obj,
                                                   gpu_FPMcost);
      FPMcost_dirtyOnGpu = true;
    }
  }
  emxFree_real_T(&b_x);
  hipMemcpy(*gpu_pu, pu, 336000ULL, hipMemcpyHostToDevice);
  EvaluationPath_GPU_kernel19<<<dim3(40U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_pu, *gpu_y);
  EvaluationPath_GPU_kernel20<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_y, *b_gpu_y);
  hipMemcpy(gpu_obj, obj, 1008072ULL, hipMemcpyHostToDevice);
  EvaluationPath_GPU_kernel21<<<dim3(40U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gpu_obj, *gpu_pu, *gpu_a);
  EvaluationPath_GPU_kernel22<<<dim3(40U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_a, *gpu_y);
  EvaluationPath_GPU_kernel23<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_y, *c_gpu_y);
  EvaluationPath_GPU_kernel24<<<dim3(40U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_pu, *gpu_a);
  EvaluationPath_GPU_kernel25<<<dim3(40U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_a, *gpu_y);
  EvaluationPath_GPU_kernel26<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      *gpu_y, *gpu_dv);
  if (FPMcost->size[1] == 1) {
    b_obj = static_cast<int32_T>(obj->NP);
  } else {
    b_obj = FPMcost->size[1];
  }
  if ((FPMcost->size[1] == static_cast<int32_T>(obj->NP)) &&
      (b_obj == static_cast<int32_T>(obj->NP))) {
    scale = obj->R[1];
    absxk = obj->R[0];
    b_iv[1] = static_cast<int32_T>(obj->NP);
    i = pw->size[0] * pw->size[1];
    pw->size[0] = 1;
    pw->size[1] = static_cast<int32_T>(obj->NP);
    emxEnsureCapacity_real_T(pw, i);
    xneg = mwGetLaunchParameters1D(
        static_cast<real_T>((static_cast<int32_T>(obj->NP) - 1) + 1LL), &grid,
        &block, 1024U, 65535U);
    if (xneg) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_pw, pw);
      EvaluationPath_GPU_kernel27<<<grid, block>>>(
          gpu_obj, *gpu_dv, absxk, *c_gpu_y, scale, *b_gpu_y, gpu_FPMcost,
          b_iv[1] - 1, gpu_pw);
      pw_dirtyOnGpu = true;
    }
  } else {
    if (FPMcost_dirtyOnGpu) {
      gpuEmxMemcpyGpuToCpu_real_T(FPMcost, &gpu_FPMcost);
    }
    hipMemcpy(y, *b_gpu_y, 8000ULL, hipMemcpyDeviceToHost);
    hipMemcpy(b_y, *c_gpu_y, 8000ULL, hipMemcpyDeviceToHost);
    hipMemcpy(dv, *gpu_dv, 8000ULL, hipMemcpyDeviceToHost);
    binary_expand_op(pw, FPMcost, y, obj, b_y, dv);
  }
  emxFree_real_T(&FPMcost);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (pw_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real_T(pw, &gpu_pw);
  }
  if (remove_sample_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_boolean_T(remove_sample, &gpu_remove_sample);
  }
  gpuEmxFree_real_T(&gpu_grade);
  gpuEmxFree_boolean_T(&gpu_remove_sample);
  gpuEmxFree_real_T(&gpu_grade_temp);
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_FPMcost);
  hipFree(*gpu_pu);
  hipFree(*gpu_y);
  hipFree(*b_gpu_y);
  hipFree(gpu_obj);
  hipFree(*gpu_a);
  hipFree(*c_gpu_y);
  hipFree(*gpu_dv);
  gpuEmxFree_real_T(&gpu_th_min);
  gpuEmxFree_real_T(&gpu_th_max);
  gpuEmxFree_real_T(&gpu_grade_obs);
  gpuEmxFree_real_T(&b_gpu_a);
  gpuEmxFree_real_T(&gpu_phaiR);
  gpuEmxFree_real_T(&gpu_PoQo);
  gpuEmxFree_real_T(&gpu_distance_obs);
  gpuEmxFree_real_T(&gpu_pw);
  hipFree(*gpu_state);
  hipFree(*gpu_preobs);
  hipFree(*b_gpu_x);
  gpuEmxFree_boolean_T(&c_gpu_x);
}

// End of code generation (EvaluationPath_GPU.cu)
