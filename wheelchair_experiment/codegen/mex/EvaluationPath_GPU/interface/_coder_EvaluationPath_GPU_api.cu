//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_EvaluationPath_GPU_api.cu
//
// Code generation for function '_coder_EvaluationPath_GPU_api'
//

// Include files
#include "_coder_EvaluationPath_GPU_api.h"
#include "EvaluationPath_GPU.h"
#include "EvaluationPath_GPU_data.h"
#include "EvaluationPath_GPU_emxutil.h"
#include "EvaluationPath_GPU_types.h"
#include "rt_nonfinite.h"

// Variable Definitions
static const int32_T iv[3]{2, 21, 1000};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[84000]);

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[63000];

static real_T (*b_emlrt_marshallIn(const mxArray *pu,
                                   const char_T *identifier))[42000];

static void c_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[42000]);

static real_T (*c_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[42000];

static real_T d_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

static void d_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[2]);

static void e_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[84000]);

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[63000];

static void emlrt_marshallIn(const mxArray *obj, const char_T *identifier,
                             struct0_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct0_T *y);

static real_T emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId);

static real_T (*emlrt_marshallIn(const mxArray *px,
                                 const char_T *identifier))[63000];

static void emlrt_marshallIn(const mxArray *preobs, const char_T *identifier,
                             cell_wrap_0 y[22]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             cell_wrap_0 y[22]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, real_T y[2]);

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u);

static const mxArray *emlrt_marshallOut(const emxArray_boolean_T *u);

static void f_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[42000]);

static real_T (*f_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[42000];

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[84000])
{
  e_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static real_T (*b_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[63000]
{
  real_T(*y)[63000];
  y = e_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*b_emlrt_marshallIn(const mxArray *pu,
                                   const char_T *identifier))[42000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[42000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = c_emlrt_marshallIn(emlrtAlias(pu), &thisId);
  emlrtDestroyArray(&pu);
  return y;
}

static void c_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[42000])
{
  f_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static real_T (*c_emlrt_marshallIn(const mxArray *u,
                                   const emlrtMsgIdentifier *parentId))[42000]
{
  real_T(*y)[42000];
  y = f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T d_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 0U, (void *)&dims);
  ret = *(real_T *)emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static void d_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[2])
{
  static const int32_T dims[2]{1, 2};
  real_T(*r)[2];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[2])emlrtMxGetData(src);
  ret[0] = (*r)[0];
  ret[1] = (*r)[1];
  emlrtDestroyArray(&src);
}

static void e_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[84000])
{
  static const int32_T dims[3]{4, 21, 1000};
  real_T(*r)[84000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 3U,
                          (void *)&dims[0]);
  r = (real_T(*)[84000])emlrtMxGetData(src);
  for (int32_T i{0}; i < 84000; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static real_T (*e_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[63000]
{
  static const int32_T dims[3]{3, 21, 1000};
  real_T(*ret)[63000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 3U,
                          (void *)&dims[0]);
  ret = (real_T(*)[63000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static void emlrt_marshallIn(const mxArray *obj, const char_T *identifier,
                             struct0_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(obj), &thisId, y);
  emlrtDestroyArray(&obj);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct0_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[10]{"K", "NP", "ObsNum", "r_wheel", "r_obs",
                                      "R", "W",  "Vref",   "limit",   "target"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 10,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "K";
  y->K =
      emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0,
                                                      0, (const char_T *)"K")),
                       &thisId);
  thisId.fIdentifier = "NP";
  y->NP =
      emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0,
                                                      1, (const char_T *)"NP")),
                       &thisId);
  thisId.fIdentifier = "ObsNum";
  y->ObsNum = emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                     (const char_T *)"ObsNum")),
      &thisId);
  thisId.fIdentifier = "r_wheel";
  y->r_wheel = emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 3,
                                     (const char_T *)"r_wheel")),
      &thisId);
  thisId.fIdentifier = "r_obs";
  y->r_obs = emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                     (const char_T *)"r_obs")),
      &thisId);
  thisId.fIdentifier = "R";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                                  (const char_T *)"R")),
                   &thisId, y->R);
  thisId.fIdentifier = "W";
  y->W =
      emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0,
                                                      6, (const char_T *)"W")),
                       &thisId);
  thisId.fIdentifier = "Vref";
  y->Vref = emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 7,
                                     (const char_T *)"Vref")),
      &thisId);
  thisId.fIdentifier = "limit";
  b_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 8,
                                                    (const char_T *)"limit")),
                     &thisId, y->limit);
  thisId.fIdentifier = "target";
  c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 9,
                                                    (const char_T *)"target")),
                     &thisId, y->target);
  emlrtDestroyArray(&u);
}

static real_T emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*emlrt_marshallIn(const mxArray *px,
                                 const char_T *identifier))[63000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[63000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(px), &thisId);
  emlrtDestroyArray(&px);
  return y;
}

static void emlrt_marshallIn(const mxArray *preobs, const char_T *identifier,
                             cell_wrap_0 y[22])
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(preobs), &thisId, y);
  emlrtDestroyArray(&preobs);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             cell_wrap_0 y[22])
{
  emlrtMsgIdentifier thisId;
  int32_T b_iv[1];
  char_T str[11];
  boolean_T bv[1];
  thisId.fParent = parentId;
  thisId.bParentIsCell = true;
  bv[0] = false;
  b_iv[0] = 22;
  emlrtCheckCell(emlrtRootTLSGlobal, parentId, u, 1U, &b_iv[0], &bv[0]);
  for (int32_T i{0}; i < 22; i++) {
    sprintf(&str[0], "%d", i + 1);
    thisId.fIdentifier = &str[0];
    emlrt_marshallIn(
        emlrtAlias(emlrtGetCell(emlrtRootTLSGlobal, parentId, u, i)), &thisId,
        y[i].f1);
  }
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, real_T y[2])
{
  d_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static const mxArray *emlrt_marshallOut(const emxArray_real_T *u)
{
  static const int32_T b_iv[2]{0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateNumericArray(2, (const void *)&b_iv[0], mxDOUBLE_CLASS,
                              mxREAL);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 2);
  emlrtAssign(&y, m);
  return y;
}

static const mxArray *emlrt_marshallOut(const emxArray_boolean_T *u)
{
  static const int32_T b_iv[2]{0, 0};
  const mxArray *m;
  const mxArray *y;
  y = nullptr;
  m = emlrtCreateLogicalArray(2, &b_iv[0]);
  emlrtMxSetData((mxArray *)m, &u->data[0]);
  emlrtSetDimensions((mxArray *)m, &u->size[0], 2);
  emlrtAssign(&y, m);
  return y;
}

static void f_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[42000])
{
  real_T(*r)[42000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 3U, (void *)&iv[0]);
  r = (real_T(*)[42000])emlrtMxGetData(src);
  for (int32_T i{0}; i < 42000; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static real_T (*f_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[42000]
{
  real_T(*ret)[42000];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 3U, (void *)&iv[0]);
  ret = (real_T(*)[42000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

void b_EvaluationPath_GPU_api(EvaluationPath_GPUStackData *SD,
                              const mxArray *const prhs[4], int32_T nlhs,
                              const mxArray *plhs[2])
{
  cell_wrap_0 preobs[22];
  emxArray_boolean_T *remove_sample;
  emxArray_real_T *pw;
  real_T(*px)[63000];
  real_T(*pu)[42000];
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&pw, 2, true);
  emxInit_boolean_T(&remove_sample, 2, true);
  // Marshall function inputs
  emlrt_marshallIn(emlrtAliasP(prhs[0]), "obj", &SD->f0.obj);
  px = emlrt_marshallIn(emlrtAlias(prhs[1]), "px");
  pu = b_emlrt_marshallIn(emlrtAlias(prhs[2]), "pu");
  emlrt_marshallIn(emlrtAliasP(prhs[3]), "preobs", preobs);
  // Invoke the target function
  EvaluationPath_GPU(&SD->f0.obj, *px, *pu, preobs, pw, remove_sample);
  // Marshall function outputs
  pw->canFreeData = false;
  plhs[0] = emlrt_marshallOut(pw);
  emxFree_real_T(&pw);
  if (nlhs > 1) {
    remove_sample->canFreeData = false;
    plhs[1] = emlrt_marshallOut(remove_sample);
  }
  emxFree_boolean_T(&remove_sample);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

// End of code generation (_coder_EvaluationPath_GPU_api.cu)
