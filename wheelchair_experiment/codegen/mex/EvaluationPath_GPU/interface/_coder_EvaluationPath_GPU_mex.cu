//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_EvaluationPath_GPU_mex.cu
//
// Code generation for function '_coder_EvaluationPath_GPU_mex'
//

// Include files
#include "_coder_EvaluationPath_GPU_mex.h"
#include "EvaluationPath_GPU_data.h"
#include "EvaluationPath_GPU_initialize.h"
#include "EvaluationPath_GPU_terminate.h"
#include "EvaluationPath_GPU_types.h"
#include "_coder_EvaluationPath_GPU_api.h"
#include "rt_nonfinite.h"

// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  EvaluationPath_GPUStackData *c_EvaluationPath_GPUStackDataGl{nullptr};
  c_EvaluationPath_GPUStackDataGl = new EvaluationPath_GPUStackData;
  mexAtExit(&EvaluationPath_GPU_atexit);
  // Module initialization.
  EvaluationPath_GPU_initialize();
  // Dispatch the entry-point.
  unsafe_EvaluationPath_GPU_mexFunction(c_EvaluationPath_GPUStackDataGl, nlhs,
                                        plhs, nrhs, prhs);
  // Module termination.
  EvaluationPath_GPU_terminate();
  delete c_EvaluationPath_GPUStackDataGl;
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2021a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           nullptr);
  return emlrtRootTLSGlobal;
}

void unsafe_EvaluationPath_GPU_mexFunction(EvaluationPath_GPUStackData *SD,
                                           int32_T nlhs, mxArray *plhs[2],
                                           int32_T nrhs, const mxArray *prhs[4])
{
  const mxArray *outputs[2];
  int32_T b_nlhs;
  // Check for proper number of arguments.
  if (nrhs != 4) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 4, 4, 18, "EvaluationPath_GPU");
  }
  if (nlhs > 2) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 18,
                        "EvaluationPath_GPU");
  }
  // Call the function.
  b_EvaluationPath_GPU_api(SD, prhs, nlhs, outputs);
  // Copy over outputs to the caller.
  if (nlhs < 1) {
    b_nlhs = 1;
  } else {
    b_nlhs = nlhs;
  }
  emlrtReturnArrays(b_nlhs, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_EvaluationPath_GPU_mex.cu)
